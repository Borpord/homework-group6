#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
using namespace std;
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <cmath>
#include <inttypes.h>
#include <gmp.h>
#include "./MSM.cuh"
#define SIZE 4

template<uint32_t limbs>
__global__ void kernel(uint32_t* d_a, uint32_t* d_b, uint32_t* d_c){
	np0=computeNP0(d_c[0]);
	uint64_t evenOdd[limbs];
	bool carry=mp_mul_red_cl<limbs>(evenOdd,d_a,d_b,d_c);
	mp_merge_cl<limbs>(d_a,evenOdd,carry);
}

int main() {
	// 在主机内存中定义输入数据
	uint32_t a[SIZE] = { 0x01345670, 0xabcdef12, 0x87654321, 0x0edcba10 };
	uint32_t b[SIZE] = { 0x01345670, 0xabcdef21, 0x87654398, 0x0edcba10 };
	uint32_t c[SIZE] = { 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff };
	uint32_t r[SIZE];
	uint32_t r_[SIZE];
	
	mpz_t A, B, C, R;
	size_t size = sizeof(a) / sizeof(uint32_t);

	mpz_init(A); mpz_init(B); mpz_init(C); mpz_init(R);
	mpz_import(A, size, -1, sizeof(uint32_t), 0, 0, a);
	mpz_import(B, size, -1, sizeof(uint32_t), 0, 0, b);
	mpz_import(C, size, -1, sizeof(uint32_t), 0, 0, c);
	mpz_mul(R, A, B);
	mpz_mod(R, R, C);
	gmp_printf("Result: %Zd\n", R);
	size_t mark;
	mpz_export(r, &mark, -1, sizeof(uint32_t), 0, 0, R);
	while (mark<SIZE)
		r[mark++] = 0;

	cout << "GMP:\n";
	//cout<<"MARK:"<<mark<<endl;
	for (int i = 0; i<SIZE; i++)
	{
		cout << hex << r[i] << endl;
	}
	
	uint32_t* d_a;
	uint32_t* d_b;
	uint32_t* d_c;
	hipMalloc(&d_a, sizeof(uint32_t)*SIZE);
	hipMalloc(&d_b, sizeof(uint32_t)*SIZE);
	hipMalloc(&d_c, sizeof(uint32_t)*SIZE);
	hipMemcpy(d_a, a, sizeof(uint32_t)*SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(uint32_t)*SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, sizeof(uint32_t)*SIZE, hipMemcpyHostToDevice);

	kernel<SIZE> <<<1,1>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();
	hipMemcpy(r_, d_a, sizeof(uint32_t)*SIZE, hipMemcpyDeviceToHost);

	cout << "test:\n";
	for (int i = 0; i<SIZE; i++)
	{
		cout << hex << r_[i] << endl;
	}
	return 0;
}
