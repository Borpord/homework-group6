#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
using namespace std;
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <cmath>
#include <inttypes.h>
#include <gmp.h>
#include "./bigint.cuh"
#define SIZE 8

__device__ void copyArrayValues(uint32_t* dest, uint32_t* src, uint32_t size) {
    for (int i = 0; i < size; i++) {
        dest[i] = src[i];
    }
}

__device__ uint32_t mod[SIZE] = { 
    0xf0000001, 0x43e1f593, 
    0x79b97091, 0x2833e848, 
    0x8181585d, 0xb85045b6, 
    0xe131a029, 0x30644e72};

__device__ const uint32_t MODULUS_[SIZE+1] = { 
	0xf0000001, 0x43e1f593, 
	0x79b97091, 0x2833e848, 
	0x8181585d, 0xb85045b6, 
	0xe131a029, 0x30644e72, 0x0};

template<uint32_t limbs>
__global__ void kernel(uint32_t* d_a, uint32_t* d_b, uint32_t* d_c){
	/*
	np0=computeNP0(mod[0]);

	uint32_t a[limbs]={},b[limbs]={};
	copyArrayValues(a,d_a,limbs);
	copyArrayValues(b,d_b,limbs);	
	
    uint64_t evenOdd[limbs]={};
	bool carry=mp_mul_red_cl<limbs>(evenOdd,a,b,mod);
	mp_merge_cl<limbs>(d_c,evenOdd,carry);*/

	/*
	uint32_t a[limbs+1]={},b[limbs+1]={},c[limbs+1]={};
    copyArrayValues(a,d_a,limbs);
    copyArrayValues(b,(uint32_t*)d_b,limbs);
    
    mp_add<limbs+1>(c,a,b);
    if(mp_comp_ge<limbs+1>(c,MODULUS_)){
        mp_sub<limbs+1>(c,c,MODULUS_);
    }
    copyArrayValues(d_c,c,limbs);*/
	
	uint32_t a[limbs+1]={},b[limbs+1]={},c[limbs+1]={};
    copyArrayValues(a,d_a,limbs);
    copyArrayValues(b,(uint32_t*)d_b,limbs);

    mp_sub<limbs+1>(c,a,b);
    if(mp_comp_gt<limbs+1>(c,a)){
        mp_add<limbs+1>(c,c,MODULUS_);
    }
    copyArrayValues(d_c,c,limbs);
}

void to_mont(uint32_t* data, mpz_t mp_bigint, mpz_t MOD){
	uint32_t temp[SIZE+1]={};
	temp[SIZE]=0x00000001;

	mpz_t mpz_temp;
	mpz_init(mpz_temp);
	mpz_import(mpz_temp, SIZE+1, -1, sizeof(uint32_t), 0, 0, temp);

	mpz_mul(mp_bigint,mp_bigint,mpz_temp);
	mpz_mod(mp_bigint,mp_bigint,MOD);
	size_t mark;
	mpz_export(data, &mark, -1, sizeof(uint32_t), 0, 0, mp_bigint);
	while (mark<SIZE)
		data[mark++] = 0;

}

void mont_back(uint32_t* data, mpz_t mp_bigint, mpz_t MOD){
	uint32_t temp[SIZE+1]={};
	temp[SIZE]=0x00000001;

	mpz_t mpz_temp;
	mpz_init(mpz_temp);
	mpz_import(mpz_temp, SIZE+1, -1, sizeof(uint32_t), 0, 0, temp);
	
	mpz_invert(mpz_temp,mpz_temp,MOD);
	mpz_mul(mp_bigint,mp_bigint,mpz_temp);
	mpz_mod(mp_bigint,mp_bigint,MOD);
	size_t mark;
	mpz_export(data, &mark, -1, sizeof(uint32_t), 0, 0, mp_bigint);
	while (mark<SIZE)
		data[mark++] = 0;

}

int main() {
	// 在主机内存中定义输入数据
	uint32_t a[SIZE];
	uint32_t b[SIZE];
	for(size_t count=0;count<1000000;count++){
		srand(count);
		for(size_t j=0;j<SIZE;j++){
			a[j]=(rand()<<16)|(rand());
			b[j]=(rand()<<16)|(rand());
		}

		uint32_t c[SIZE] = { 
		0xf0000001, 0x43e1f593, 
		0x79b97091, 0x2833e848, 
		0x8181585d, 0xb85045b6, 
		0xe131a029, 0x30644e72};
		uint32_t r[SIZE];
		uint32_t r_[SIZE];
	
		mpz_t A, B, C, R;

		mpz_init(A); mpz_init(B); mpz_init(C); mpz_init(R);
		mpz_import(A, SIZE, -1, sizeof(uint32_t), 0, 0, a);
		mpz_import(B, SIZE, -1, sizeof(uint32_t), 0, 0, b);
		mpz_import(C, SIZE, -1, sizeof(uint32_t), 0, 0, c);
		mpz_sub(R, A, B);
		mpz_mod(R, R, C);
		//gmp_printf("Result: %Zd\n", R);
		size_t mark;
		mpz_export(r, &mark, -1, sizeof(uint32_t), 0, 0, R);
		while (mark<SIZE)
			r[mark++] = 0;
		/*
		cout << "GMP:\n";
		//cout<<"MARK:"<<mark<<endl;
		for (int i = 0; i<SIZE; i++)
		{
			cout << hex << r[i] << endl;
		}*/
	
		to_mont(a,A,C);
		to_mont(b,B,C);
	
		uint32_t* d_a;
		uint32_t* d_b;
		uint32_t* d_c;
		hipMalloc(&d_a, sizeof(uint32_t)*SIZE);
		hipMalloc(&d_b, sizeof(uint32_t)*SIZE);
		hipMalloc(&d_c, sizeof(uint32_t)*SIZE);
		hipMemcpy(d_a, a, sizeof(uint32_t)*SIZE, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, sizeof(uint32_t)*SIZE, hipMemcpyHostToDevice);

		kernel<SIZE> <<<1,1>>>(d_a, d_b, d_c);
		hipDeviceSynchronize();
		hipMemcpy(r_, d_c, sizeof(uint32_t)*SIZE, hipMemcpyDeviceToHost);

		mpz_import(A, SIZE, -1, sizeof(uint32_t), 0, 0, r_);
		mont_back(r_,A,C);
		/*
		cout << "test:\n";
		for (int i = 0; i<SIZE; i++)
		{
			cout << hex << r_[i] << endl;
		}*/
		for (int i = 0; i<SIZE; i++)
		{
			if(r_[i]!=r[i]){
				printf("error!\n");
				return -1;
			}
		}
	}
	
	printf("success!\n");
	return 0;
}
